#include "hip/hip_runtime.h"
#include <stdio.h>

#include "ChangeVertices_cuda.h"




 __global__ void change_vertices(float3 * outVertices,float3 * originVertices , float3 changeModelCenter, int verticesSize, float changeTime, int width, int height) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const float scaleValue = abs(cos(changeTime)) * 0.6;

	if (idx < verticesSize) {
		const float3 originVertice = originVertices[idx];
		const float3 changeDir = changeModelCenter - originVertice;
		const float3 changeVertice = outVertices[idx];
		outVertices[idx] = originVertice + changeDir * scaleValue;

		//if (idx == 100000) {
		//	printf("idx: %d , vertice: %f %f %f \n", idx, changeVertice.x, changeVertice.y, changeVertice.z);
		//	printf("time: %f,scale: %f, changeDir: %f %f %f \n", changeTime, scaleValue, changeDir.x, changeDir.y, changeDir.z);
		//}
	}
}






void changeVerticesPos(float3 * outVertices, float3* originVertices, float3 changeModelCenter, int verticesSize, float changeTime) {

	const int blockMaxSize = 512;
	int threadSize = verticesSize % blockMaxSize == 0 ? verticesSize / blockMaxSize : (verticesSize / blockMaxSize + 1);
	dim3 threadsPerBlock(threadSize, 1);
	change_vertices << <blockMaxSize, threadsPerBlock >> > (outVertices, originVertices, changeModelCenter, verticesSize, changeTime, blockMaxSize, threadSize);
}